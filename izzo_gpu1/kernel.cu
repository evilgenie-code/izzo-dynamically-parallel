#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cmath>
#include <cstdio>
#include <cstdlib> 
#include <ctime>



void vers(const double* vIn, double* verOut)
{
    double vMod = 0;

    for (int i = 0; i < 3; i++)
    {
        vMod += vIn[i] * vIn[i];
    }

    double sqrtVMod = sqrt(vMod);

    for (int i = 0; i < 3; i++)
    {
        verOut[i] = vIn[i] / sqrtVMod;
    }
}

void vett(const double* vet1, const double* vet2, double* prod)
{
    prod[0] = (vet1[1] * vet2[2] - vet1[2] * vet2[1]);
    prod[1] = (vet1[2] * vet2[0] - vet1[0] * vet2[2]);
    prod[2] = (vet1[0] * vet2[1] - vet1[1] * vet2[0]);
}

__device__ double x2tof(double x, double s, double c, int lw, int m)
{
    //printf("inn = %f, s = %f, c = %f, lw = %i, revs = %i \n", x, s, c, lw, m);
	
    double am, a, alfa, beta;

    am = s / 2;
    a = am / (1 - x * x);

    if (x < 1)//ellpise
    {
        beta = 2 * asin(sqrt((s - c) / (2 * a)));
        if (lw) beta = -beta;
        alfa = 2 * acos(x);
    }
    else
    {
        alfa = 2 * acosh(x);
        beta = 2 * asinh(sqrt((s - c) / (-2 * a)));
        if (lw) beta = -beta;
    }

    if (a > 0)
    {
        return (a * sqrt(a) * ((alfa - sin(alfa)) - (beta - sin(beta)) + 2 * acos(-1.0) * m));
    }
    else
    {
        return (-a * sqrt(-a) * ((sinh(alfa) - alfa) - (sinh(beta) - beta)));
    }
}

__global__ void xf(float *inn, float  s, float c, int lw, int revs, float t, float* xF)
{
	
    int blockIndex = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x;

    int ThreadIndex = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;

    int tid = blockIndex * blockDim.x * blockDim.y * blockDim.z + ThreadIndex;
	
    float xf = log(x2tof(inn[tid], s, c, lw, revs)) - logf(t);
	
    xF[tid] = xf;
}

void getXF(float *inn, double s, double c, int lw, int revs, float t, float *xF, int iterate)
{
    float *dev_inn, *dev_xf;

    hipMalloc((void**)&dev_xf, 2 * sizeof(float));
    hipMalloc((void**)&dev_inn, 2 * sizeof(float));
    hipMemcpy(dev_xf, xF, 2 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_inn, inn, 2 * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
	
    xf << <1, 2 >> > (dev_inn, s, c, lw, revs, t, dev_xf);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
	
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("3.%i iterate %i time spent executing by the GPU: %.16e milliseconds\n", iterate, iterate, gpuTime);

    hipMemcpy(xF, dev_xf, 2 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_xf);
}

void lambert(const float* r0, const float* rk, float t, int lw, int revs, float mu)
{
    double v1[3], v2[3], r1[3], r2[3], r2Vers[3];
    double	V, T, r2Mod = 0.0,    // R2 module
        dotProd = 0.0, // dot product
        c,		        // non-dimensional chord
        s,		        // non dimesnional semi-perimeter
        am,		        // minimum energy ellipse semi major axis
        lambda,	        //lambda parameter defined in Battin's Book
        x, x1, x2, x1F, x2F, xNew = 0, yNew, err, alfa, beta, psi, eta, eta2, sigma1, vr1, vt1, vt2, vr2, r1Mod = 0.0;
    int iterate, i, leftbranch = 0;
    const double tolerance = 1e-7;
    double ihDum[3], ih[3], dum[3];

    double a, p, theta;

    if (t <= 0)
    {
        return;
    }

    for (i = 0; i < 3; i++)
    {
        r1[i] = r0[i];
        r2[i] = rk[i];
        r1Mod += r1[i] * r1[i];
    }

    r1Mod = sqrt(r1Mod);
    V = sqrt(mu / r1Mod);
    T = r1Mod / V;

    t /= T;

    for (i = 0; i < 3; i++)
    {
        r1[i] /= r1Mod;
        r2[i] /= r1Mod;
        r2Mod += r2[i] * r2[i];
    }

    r2Mod = sqrt(r2Mod);

    for (i = 0; i < 3; i++)
        dotProd += (r1[i] * r2[i]);

    theta = acos(dotProd / r2Mod);

    if (lw)
    {
        theta = 2 * acos(-1.0) - theta;
    }

    c = sqrt(1 + r2Mod * (r2Mod - 2.0 * cos(theta)));
    s = (1 + r2Mod + c) / 2.0;
    am = s / 2.0;
    lambda = sqrt(r2Mod) * cos(theta / 2.0) / s;

    float inn1, inn2;

    xNew = 0;
    iterate = 0;
    float xF[2];

    if (revs == 0)
    {
        x1 = log(0.4767);
        x2 = log(1.5233);
        inn1 = -.5233;
        inn2 = .5233;

        
        // Newton iterations
        while (fabs(x1 - xNew) > tolerance)
        //while (iterate <= 4)
        {
            iterate++;

            float inn[2] = { inn1, inn2 };
        	
            getXF(inn, s, c, lw, revs, t, xF, iterate);

            xNew = (x1 * xF[1] - x2 * xF[0]) / (xF[1] - xF[0]);

            x1 = x2;
            x2 = xNew;

            inn1 = exp(x1) - 1;
            inn2 = exp(xNew) - 1;
        }


        x = exp(xNew) - 1;
    }
    else
    {
        if (leftbranch == 1)   // left branch
        {
            inn1 = -0.5234;
            inn2 = -0.2234;
        }
        else			   // right branch
        {
            inn1 = 0.7234;
            inn2 = 0.5234;
        }

        x1 = tan(inn1 * acos(-1.0) / 2);
        x2 = tan(inn2 * acos(-1.0) / 2);
        //x1F = x2tof(inn1, s, c, lw, revs) - t;
       // x2F = x2tof(inn2, s, c, lw, revs) - t;

        int imax = 30;
        // Newton Iteration
        while ((err > tolerance) && (x1F != x2F) && iterate < imax)
        {
             iterate++;

            float inn[2] = { inn1, inn2 };
        	
            getXF(inn, s, c, lw, revs, t, xF , iterate);

            xNew = (x1 * xF[1] - x2 * xF[0]) / (xF[1] - xF[0]);

            printf("iterate = %i, fabs = %.e \n", iterate, fabs(x1 - xNew));

            x1 = x2;
            x2 = xNew;

            inn1 = exp(x1) - 1;
            inn2 = exp(xNew) - 1;
        }

        x = atan(xNew) * 2 / acos(-1.0);

        iterate = iterate == imax ? iterate - 1 : iterate;
    }
 

    a = am / (1 - x * x);		    // solution semimajor axis
    // psi evaluation
    if (x < 1)                         // ellipse
    {
        beta = 2 * asin(sqrt((s - c) / (2 * a)));
        if (lw) beta = -beta;
        alfa = 2 * acos(x);
        psi = (alfa - beta) / 2;
        eta2 = 2 * a * pow(sin(psi), 2) / s;
        eta = sqrt(eta2);
    }
    else       // hyperbola
    {
        beta = 2 * asinh(sqrt((c - s) / (2 * a)));
        if (lw) beta = -beta;
        alfa = 2 * acosh(x);
        psi = (alfa - beta) / 2;
        eta2 = -2 * a * pow(sinh(psi), 2) / s;
        eta = sqrt(eta2);
    }

    p = (r2Mod / (am * eta2)) * pow(sin(theta / 2), 2);
    sigma1 = (1 / (eta * sqrt(am))) * (2 * lambda * am - (lambda + x * eta));
    vett(r1, r2, ihDum);
    vers(ihDum, ih);

    if (lw)
    {
        for (i = 0; i < 3; i++)
            ih[i] = -ih[i];
    }

    vr1 = sigma1;
    vt1 = sqrt(p);
    vett(ih, r1, dum);

    for (i = 0; i < 3; i++)
        v1[i] = vr1 * r1[i] + vt1 * dum[i];

    vt2 = vt1 / r2Mod;
    vr2 = -vr1 + (vt1 - vt2) / tan(theta / 2);

    vers(r2, r2Vers);
    vett(ih, r2Vers, dum);
    for (i = 0; i < 3; i++)
        v2[i] = vr2 * r2[i] / r2Mod + vt2 * dum[i];

    for (i = 0; i < 3; i++)
    {
        v1[i] *= V;
        v2[i] *= V;
    }
}

int main() {

    printf("1. start program \n");
    double AU = 1.49597870691e8;
    double fMSun = 1.32712440018e11;             // km^3/sec^2

    double UnitR = AU;
    double UnitV = sqrt(fMSun / UnitR);          // km/sec
    double UnitT = (UnitR / UnitV) / 86400;         // day

    float unitT = 100.0 / UnitT;
    float mu = 1.0;
    int lw = 0, revs = 0.0;
    float r1[3] = { -7.8941608095246896e-01, -6.2501194900473045e-01, 3.5441335698377735e-05 };
    float r2[3] = { 1.3897892184188783e+00, 1.3377137029002054e-01, -3.1287386211010106e-02 };

	printf("2. calculate program \n");
	
    lambert(r1, r2, unitT, lw, revs, mu);

    printf("4. finish program \n");
}